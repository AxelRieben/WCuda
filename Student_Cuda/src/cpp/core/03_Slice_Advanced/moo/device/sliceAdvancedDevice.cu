#include "hip/hip_runtime.h"
#include "Indice1D.h"
#include "cudaTools.h"
#include "reductionADD.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

__device__ float fpiAdvanced(float x);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void sliceAdvancedDevice(float* ptrResultGM, int n);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void sliceAdvancedDevice(float* ptrResultGM, int n)
    {
    //Shared Memory
    extern __shared__ float tabSM[];

    const int NB_THREAD = Indice1D::nbThread();
    const int TID = Indice1D::tid();
    const int TID_LOCAL = Indice1D::tidLocal();
    int s = TID;

    const float DX = 1 / (float) n;
    float sommeThread = 0;
    float xs = 0;

    while (s < n)
	{
	xs = s * DX;
	sommeThread += fpiAdvanced(xs);
	s += NB_THREAD;
	}

    tabSM[TID_LOCAL] = sommeThread;

    __syncthreads();

    reductionADD(tabSM, ptrResultGM);

    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ float fpiAdvanced(float x)
    {
    return 4 / (1 + x * x);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/


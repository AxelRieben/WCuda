#include "hip/hip_runtime.h"
#include "Device.h"
#include "SliceAdvanced.h"
#include "cudaTools.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void sliceAdvancedDevice(float* ptrResultGM, int n);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

SliceAdvanced::SliceAdvanced(Grid& grid, int n) :
	grid(grid), n(n)
    {
    this->pi = 0;

    this->sizeTabSM = grid.threadCounts() * sizeof(float);

    Device::malloc(&ptrResultGM, sizeof(float)); //resultat
    Device::memclear(ptrResultGM, sizeof(float));

    Device::lastCudaError("MM (end allocation)"); // temp debug, facultatif
    }

SliceAdvanced::~SliceAdvanced()
    {
    Device::free(ptrResultGM);
    }

void SliceAdvanced::run()
    {
    Device::lastCudaError("Slice (before)");

    sliceAdvancedDevice<<<grid.dg,grid.db,sizeTabSM>>>(ptrResultGM,n);

    Device::lastCudaError("Slice (after)");

    Device::memcpyDToH(&pi, ptrResultGM, sizeof(float));

    this->pi = this->pi / n;
    }

float SliceAdvanced::getPI()
    {
    return this->pi;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/


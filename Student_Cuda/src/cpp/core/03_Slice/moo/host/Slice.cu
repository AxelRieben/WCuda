#include "hip/hip_runtime.h"
#include "Slice.h"
#include "Device.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void sliceDevice(float* ptrTabGM, int n);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

Slice::Slice(Grid& grid, int n) : grid(grid), n(n)
    {
    this->pi = 0;

    ptrTab = new float[n];

    this->sizeTab = n * sizeof(float);
    Device::malloc(&ptrTabGM, sizeTab);
    Device::memcpyHToD(ptrTabGM, ptrTab, sizeTab);
    }

Slice::~Slice()
    {
    Device::free(ptrTabGM);
    delete[] ptrTab;
    }

void Slice::run()
    {
    Device::lastCudaError("Slice (before)");

    dim3 dg = grid.dg;
    dim3 db = grid.db;

    sliceDevice<<<dg,db>>>(ptrTabGM,n);

    Device::lastCudaError("Slice (after)");

    Device::memcpyDToH(ptrTab, ptrTabGM, sizeTab);

    reduceTab();
    }

float Slice::getPI()
    {
    return this->pi;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

void Slice::reduceTab()
    {
    float sum = 0;

#pragma omp parallel for reduction(+:sum)
    for (int i = 0; i < n; i++)
	{
	sum += ptrTab[i];
	}

    this->pi = sum * (1 / (double) n);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/


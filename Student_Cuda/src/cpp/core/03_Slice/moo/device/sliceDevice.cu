#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "Indice1D.h"
#include "cudaTools.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

__device__ double fpi(double x);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void sliceDevice(float* ptrTabGM, int n);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void sliceDevice(float* ptrTabGM, int n)
    {
    const int NB_THREAD = Indice2D::nbThread();
    const int TID = Indice2D::tid();
    int s = TID;

    const double DX = 1 / (double) n;
    double sommeThread = 0;
    double xs = 0;

    while (s < n)
	{
	xs = s * DX;
	sommeThread += fpi(xs);
	s += NB_THREAD;
	}

    ptrTabGM[TID] = sommeThread;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ double fpi(double x)
    {
    return 4 / (1 + x * x);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/


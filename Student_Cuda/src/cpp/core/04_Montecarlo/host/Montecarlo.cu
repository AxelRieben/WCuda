#include "hip/hip_runtime.h"
#include "Montecarlo.h"

#include "Device.h"
#include "cudaTools.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void montecarloDevice(float* ptrResultGM, int n);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

Montecarlo::Montecarlo(Grid& grid, int n) :
	grid(grid), n(n)
    {
    this->pi = 0;

    ptrResult = 0;

    this->sizeTabSM = n * sizeof(float);

    Device::malloc(&ptrResultGM, sizeof(float)); //resultat
    Device::memclear(ptrResultGM, sizeof(float));
    }

Montecarlo::~Montecarlo()
    {
    Device::free(ptrResultGM);
    }

void Montecarlo::run()
    {
    Device::lastCudaError("Slice (before)");

    dim3 dg = grid.dg;
    dim3 db = grid.db;

    montecarloDevice<<<dg,db,sizeTabSM>>>(ptrResultGM,n);

    Device::lastCudaError("Slice (after)");

    Device::memcpyDToH(ptrResult, ptrResultGM, sizeof(float));

    this->pi = *ptrResult;
    }

float Montecarlo::getPI()
    {
    return this->pi;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/


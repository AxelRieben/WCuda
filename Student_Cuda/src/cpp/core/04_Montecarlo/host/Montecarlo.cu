#include "hip/hip_runtime.h"
#include "Montecarlo.h"
#include "Device.h"
#include "cudaTools.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void montecarloDevice(float* ptrResultGM, int n, hiprandState* tabDevGeneratorGM);
__global__ void createGenerator(hiprandState* tabDevGeneratorGM, int deviceId);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

Montecarlo::Montecarlo(Grid& grid, int n) :
	grid(grid), n(n)
    {
    this->pi = 0;
    this->n0 = 0;

    tabDevGenerator = new hiprandState[grid.threadCounts()];

    this->sizeTabSM = grid.threadCounts() * sizeof(float);
    this->sizeTabGenerator = grid.threadCounts() * sizeof(hiprandState);

    Device::malloc(&ptrResultGM, sizeof(float));
    Device::memclear(ptrResultGM, sizeof(float));

    Device::malloc(&tabDevGeneratorGM, sizeTabGenerator);
    Device::memclear(tabDevGeneratorGM, sizeTabGenerator);
    }

Montecarlo::~Montecarlo()
    {
    Device::free(ptrResultGM);
    Device::free(tabDevGeneratorGM);
    delete[] tabDevGenerator;
    }

void Montecarlo::run()
    {
    Device::lastCudaError("Slice (before)");

    dim3 dg = grid.dg;
    dim3 db = grid.db;

    createGenerator<<<dg, db>>>(tabDevGeneratorGM, 0);

    Device::memcpyDToH(tabDevGenerator, tabDevGeneratorGM, sizeof(float));

    int nPerThread = n / grid.threadCounts();
    montecarloDevice<<<dg,db,sizeTabSM>>>(ptrResultGM,nPerThread,tabDevGeneratorGM);

    Device::lastCudaError("Slice (after)");

    Device::memcpyDToH(&pi, ptrResultGM, sizeof(float));

    this->n0 = pi;
    this->pi = 2 * 4 * n0 / (float) n;
    }

float Montecarlo::getPI()
    {
    return this->pi;
    }

int Montecarlo::getN0()
    {
    return this->n0;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/


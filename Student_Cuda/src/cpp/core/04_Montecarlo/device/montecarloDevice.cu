#include "hip/hip_runtime.h"
#include "Indice1D.h"
#include "cudaTools.h"
#include "reductionADD.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

__device__ float f(float x);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void montecarloDevice(float* ptrResultGM, int n);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/


__global__ void montecarloDevice(float* ptrResultGM, int n)
    {
    //Shared Memory
    __shared__ extern float tabSM[];

    const int NB_THREAD = Indice1D::nbThread();
    //const int NB_THREAD_BLOCK = Indice1D::nbThreadBlock();
    const int TID = Indice1D::tid();
    const int TID_LOCAL = Indice1D::tidLocal();
    int s = TID;

    const float DX = 1 / (float) n;
    float sommeThread = 0;
    float xs = 0;

    while (s < n)
	{
	xs = s * DX;
	sommeThread += f(xs);
	s += NB_THREAD;
	}

    tabSM[TID_LOCAL] = sommeThread;

    __syncthreads();

    reductionADD(tabSM, ptrResultGM);

    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ float f(float x)
    {
    return sqrt(1-x*x);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/


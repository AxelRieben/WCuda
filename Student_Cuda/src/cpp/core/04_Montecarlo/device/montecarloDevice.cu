#include "hip/hip_runtime.h"
#include "Indice1D.h"
#include "cudaTools.h"
#include "reductionADD.h"
#include "Calibreur_GPU.h"
#include <hiprand/hiprand_kernel.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

__device__ float f(float x);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void montecarloDevice(float* ptrResultGM, int n, hiprandState* tabDevGeneratorGM);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void montecarloDevice(float* ptrResultGM, int n, hiprandState* tabDevGeneratorGM)
    {
    //Shared Memory
    __shared__ extern float tabSM[];

    //Montecarlo
    int n0 = 0;
    const int a = -1;
    const int b = 1;
    const int m = 2;

    const int TID = Indice1D::tid();
    const int TID_LOCAL = Indice1D::tidLocal();

    // Global Memory -> Register (optimization)
    hiprandState localGenerator = tabDevGeneratorGM [TID];
    float xAlea;
    float yAlea;

    for (long i = 1; i <= n; i++)
    {
    xAlea = a + (b-a) * hiprand_uniform(&localGenerator);
    yAlea = m * hiprand_uniform(&localGenerator);

    if(yAlea < f(xAlea))
	{
	    n0++;
	}

    }

    tabDevGeneratorGM[TID] = localGenerator;
    tabSM[TID_LOCAL] = n0;

    __syncthreads();

    reductionADD(tabSM, ptrResultGM);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ float f(float x)
    {
    return sqrt(1-(x*x));
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/


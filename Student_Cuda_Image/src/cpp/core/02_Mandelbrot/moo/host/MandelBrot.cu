#include "hip/hip_runtime.h"
#include "MandelBrot.h"

#include <iostream>
#include <assert.h>

#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void mandelBrot(uchar4* ptrDevPixels, uint w, uint h, uint n, DomaineMath domaineMath);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

MandelBrot::MandelBrot(const Grid& grid, uint w, uint h, int dt, uint n, const DomaineMath& domaineMath) :
	Animable_I<uchar4>(grid, w, h, "Mandelbrot_Cuda_RGBA_uchar4", domaineMath), variateurAnimation(Interval<uint>(30, n), dt)
    {
    // Inputs
    this->n = n;

    // Tools
    this->t = 0; // protected dans Animable
    }

MandelBrot::~MandelBrot()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void MandelBrot::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("mandelbrot rgba uchar4 (before kernel)"); // facultatif, for debug only, remove for release

    // le kernel est importer ci-dessus (ligne 19)
    mandelBrot<<<dg, db>>>(ptrDevPixels,w,h,n,domaineMath);

    Device::lastCudaError("mandelbrot rgba uchar4 (after kernel)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 * Call periodicly by the API
 */
void MandelBrot::animationStep()
    {
    this->t = variateurAnimation.varierAndGet();
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/


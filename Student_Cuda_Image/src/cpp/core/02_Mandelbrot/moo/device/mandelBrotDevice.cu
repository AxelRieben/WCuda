#include "hip/hip_runtime.h"
#include "MandelBrotMath.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "IndiceTools_GPU.h"
#include "DomaineMath_GPU.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void mandelBrot(uchar4* ptrDevPixels, uint w, uint h, uint n, DomaineMath domaineMath);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void mandelBrot(uchar4* ptrDevPixels, uint w, uint h, uint n, DomaineMath domaineMath)
    {
    MandelBrotMath mandelBrotMath = MandelBrotMath(n);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    double x = 0;
    double y = 0;

    int i;
    int j;

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j);

	domaineMath.toXY(i, j, &x, &y); // fill (x,y) from (i,j)

	mandelBrotMath.colorXY(&ptrDevPixels[s], x, y, n); // in [01]

	s += NB_THREAD;
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/


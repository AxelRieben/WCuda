#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"
#include "RayTracingMath.h"

#include "IndiceTools_GPU.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rayTracing(uchar4* ptrDevPixels, Sphere* ptrDevTabSphere, int nbSphere, uint w, uint h, float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rayTracing(uchar4* ptrDevPixels, Sphere* ptrDevTabSphere, int nbSphere, uint w, uint h, float t)
    {
    RayTracingMath rayTracingMath = RayTracingMath(w, h, ptrDevTabSphere, nbSphere);

    const int WH = w * h;
    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    int i;	// in [0,h[
    int j; 	// in [0,w[

    int s = TID;  // in [0,...
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j); 	// update (i, j)
	rayTracingMath.colorIJ(&ptrDevPixels[s], i, j, t); 	// update ptrDevPixels[s]
	s += NB_THREAD;
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/


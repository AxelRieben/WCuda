#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"
#include "RayTracingMath.h"

#include "IndiceTools_GPU.h"
#include "length_cm.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

// Déclaration Constante globale
__constant__ Sphere TAB_CM[LENGTH_CM];

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rayTracingSM(uchar4* ptrDevPixels, Sphere* ptrDevTabSphere, int nbSphere, uint w, uint h, float t);

__global__ void rayTracingCM(uchar4* ptrDevPixels, Sphere* ptrDevTabSphere, int nbSphere, uint w, uint h, float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ void work(uchar4* ptrDevPixels, Sphere* ptrDevTabSphere, int nbSphere, uint w, uint h, float t);

__device__ void copyGMtoSM(Sphere* tabSM, Sphere* tabGM, int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__ void uploadToCM(Sphere* ptrTabSphere)
    {
    size_t size = LENGTH_CM * sizeof(Sphere);
    int offset = 0;
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(TAB_CM), ptrTabSphere, size, offset, hipMemcpyHostToDevice));
    //Device::memcpyToCM(TAB_CM, ptrTabSphere, size); //Not working
    }

__global__ void rayTracingSM(uchar4* ptrDevPixels, Sphere* ptrDevTabSphere, int nbSphere, uint w, uint h, float t)
    {
    //Shared Memory
    __shared__ extern Sphere tabSM[];
    copyGMtoSM(tabSM, ptrDevTabSphere, nbSphere);
    __syncthreads();
    work(ptrDevPixels, tabSM, LENGTH_CM, w, h, t);
    }

__global__ void rayTracingCM(uchar4* ptrDevPixels, Sphere* ptrDevTabSphere, int nbSphere, uint w, uint h, float t)
    {
    //Constant memory
    work(ptrDevPixels, TAB_CM, LENGTH_CM, w, h, t);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ void work(uchar4* ptrDevPixels, Sphere* ptrDevTabSphere, int nbSphere, uint w, uint h, float t)
    {
    RayTracingMath rayTracingMath = RayTracingMath(w, h, ptrDevTabSphere, nbSphere);

    const int WH = w * h;
    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    int i;	// in [0,h[
    int j; 	// in [0,w[

    int s = TID;  // in [0,...
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j); 	// update (i, j)
	rayTracingMath.colorIJ(&ptrDevPixels[s], i, j, t); 	// update ptrDevPixels[s]
	s += NB_THREAD;
	}
    }

__device__ void copyGMtoSM(Sphere* tabSM, Sphere* tabGM, int n)
    {
    const int NB_THREAD_LOCAL = Indice2D::nbThreadLocal();
    const int TID_LOCAL = Indice2D::tidLocal();
    int s = TID_LOCAL;

    while(s < n)
	{
	tabSM[s] = tabGM[s];
	s += NB_THREAD_LOCAL;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/


#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"
#include "RayTracingMath.h"

#include "IndiceTools_GPU.h"
#include "length_cm.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

// Déclaration Constante globale
__constant__ Sphere TAB_CM[LENGTH_CM];

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rayTracing(uchar4* ptrDevPixels, Sphere* ptrDevTabSphere, int nbSphere, uint w, uint h, float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ void work(uchar4* ptrDevPixels, Sphere* ptrDevTabSphere, int nbSphere, uint w, uint h, float t);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__ void uploadToCM(Sphere* ptrTabSphere)
    {
    size_t size = LENGTH_CM * sizeof(Sphere);
    int offset = 0;
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(TAB_CM), ptrTabSphere, size, offset, hipMemcpyHostToDevice));
    //Device::memcpyToCM(TAB_CM, ptrTabSphere, size);
    }

__global__ void rayTracing(uchar4* ptrDevPixels, Sphere* ptrDevTabSphere, int nbSphere, uint w, uint h, float t)
    {
    work(ptrDevPixels, TAB_CM, LENGTH_CM, w, h, t);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ void work(uchar4* ptrDevPixels, Sphere* ptrDevTabSphere, int nbSphere, uint w, uint h, float t)
    {
    RayTracingMath rayTracingMath = RayTracingMath(w, h, ptrDevTabSphere, nbSphere);

    const int WH = w * h;
    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    int i;	// in [0,h[
    int j; 	// in [0,w[

    int s = TID;  // in [0,...
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j); 	// update (i, j)
	rayTracingMath.colorIJ(&ptrDevPixels[s], i, j, t); 	// update ptrDevPixels[s]
	s += NB_THREAD;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/


#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Device.h"
#include "RayTracing.h"
#include "length_cm.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void rayTracingGM(uchar4* ptrDevPixels, Sphere* ptrDevTabSphere, int nbSphere, uint w, uint h, float t);
extern __global__ void rayTracingSM(uchar4* ptrDevPixels,Sphere* ptrDevTabSphere,int nbSphere,uint w, uint h,float t);
extern __global__ void rayTracingCM(uchar4* ptrDevPixels,Sphere* ptrDevTabSphere,int nbSphere,uint w, uint h,float t);

extern void uploadToCM(Sphere* ptrTabSphere);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

RayTracing::RayTracing(const Grid& grid, uint w, uint h, float dt) :
	Animable_I<uchar4>(grid, w, h, "RayTracing_Cuda_RGBA_uchar4")
    {
    // Inputs
    this->dt = dt;

    // Tools
    this->t = 0; // protected dans Animable

    this->nbSphere = LENGTH_CM;

    // Fabrication coté host des données
    SphereCreator* creator = new SphereCreator(nbSphere, w, h);
    Sphere* ptrTabSphere = creator->getTabSphere();
    this->ptrDevTabSphere = 0;

    //Global Memory
    fillGM(ptrTabSphere);

    //Constant Memory
    //fillCM(ptrTabSphere);
    }

RayTracing::~RayTracing()
    {
    Device::free(ptrDevTabSphere);
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

__host__ void RayTracing::fillCM(Sphere* ptrTabSphere)
    {
// Appelle le service d'upload coté device
    uploadToCM(ptrTabSphere);
    }

__host__ void RayTracing::fillGM(Sphere* ptrTabSphere)
    {
    this->sizeOctetSphere = this->nbSphere * sizeof(Sphere);
    Device::malloc(&ptrDevTabSphere, sizeOctetSphere);
    Device::memcpyHToD(ptrDevTabSphere, ptrTabSphere, sizeOctetSphere);
    }

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void RayTracing::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("RayTracing rgba uchar4 (before)"); // facultatif, for debug only, remove for release

    //Global Memory
    rayTracingGM<<<dg,db>>>(ptrDevPixels,ptrDevTabSphere,nbSphere,w,h,t);

    //Constant Memory
    //rayTracingCM<<<dg,db>>>(ptrDevPixels,ptrDevTabSphere,nbSphere,w,h,t);

    //Shared Memory
    //size_t sizeOctetSM = nbSphere * sizeof(Sphere);
    //rayTracingSM<<<dg,db, sizeOctetSM>>>(ptrDevPixels,ptrDevTabSphere,nbSphere,w,h,t);

    Device::lastCudaError("RayTracing rgba uchar4 (after)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 * Call periodicly by the API
 */
void RayTracing::animationStep()
    {
    t += dt;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/


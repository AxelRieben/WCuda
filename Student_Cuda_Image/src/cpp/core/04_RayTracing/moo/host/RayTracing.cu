#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Device.h"
#include "RayTracing.h"
#include "length_cm.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void rayTracing(uchar4* ptrDevPixels,Sphere* ptrDevTabSphere,int nbSphere,uint w, uint h,float t);
extern void uploadToCM(Sphere* ptrTabSphere);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__host__ void fillCM(Sphere* ptrTabSphere);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

RayTracing::RayTracing(const Grid& grid, uint w, uint h, float dt) :
	Animable_I<uchar4>(grid, w, h, "RayTracing_Cuda_RGBA_uchar4")
    {
    // Inputs
    this->dt = dt;

    // Tools
    this->t = 0; // protected dans Animable

    // Fabrication coté host des données
    SphereCreator* creator = new SphereCreator(LENGTH_CM, w, h);
    Sphere* ptrTabSphere = creator->getTabSphere();

    this->nbSphere = 40;
    //SphereCreator* creator = new SphereCreator(nbSphere, w, h);
    //Sphere* ptrTabSphere = creator->getTabSphere();
    //this->sizeOctetSphere = nbSphere * sizeof(Sphere);
    //Device::malloc(&ptrDevTabSphere, sizeOctetSphere);
    //Device::memcpyHToD(ptrDevTabSphere, ptrTabSphere, sizeOctetSphere);

    fillCM(ptrTabSphere);

    }

RayTracing::~RayTracing()
    {
    Device::free(ptrDevTabSphere);
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

__host__ void fillCM(Sphere* ptrTabSphere)
    {
// Appelle le service d'upload coté device
    uploadToCM(ptrTabSphere);
    }

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void RayTracing::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("RayTracing rgba uchar4 (before)"); // facultatif, for debug only, remove for release

    rayTracing<<<dg,db>>>(ptrDevPixels,ptrDevTabSphere,nbSphere,w,h,t);

    Device::lastCudaError("RayTracing rgba uchar4 (after)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 * Call periodicly by the API
 */
void RayTracing::animationStep()
    {
    t += dt;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

